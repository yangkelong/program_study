
#include <hip/hip_runtime.h>
extern "C" __global__
void saxpy(int n, float a, float * x, float * y, float * out)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) out[i] = a*x[i] + y[i];
}
