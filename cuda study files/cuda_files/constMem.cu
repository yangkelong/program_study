
#include <hip/hip_runtime.h>
__constant__ int array[1024];

__global__ void kernel1(int *d_dst) {
   int tId = threadIdx.x + blockIdx.x * blockDim.x;
   d_dst[tId] = array[tId];
}

__global__ void kernel2(int *d_dst, int *d_src) {
   int tId = threadIdx.x + blockIdx.x * blockDim.x;
   d_dst[tId] = d_src[tId];
}

int main(int argc, char **argv) {
   int *d_array;
   int *d_src;
   hipMalloc((void**)&d_array, sizeof(int) * 1024);
   hipMalloc((void**)&d_src, sizeof(int) * 1024);

   int *test = new int[1024];
   memset(test, 0, sizeof(int) * 1024);

   for (int i = 0; i < 1024; i++) {
     test[i] = 100;
   }

   hipMemcpyToSymbol(HIP_SYMBOL(array), test, sizeof(int) * 1024);
   kernel1<<< 1, 1024 >>>(d_array);

   hipMemcpy(d_src, test, sizeof(int) * 1024, hipMemcpyHostToDevice);
   kernel2<<<1, 32 >>>(d_array, d_src),

   free(test);
   hipFree(d_array);
   hipFree(d_src);
   
   
   }