

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void test_shuf_broadcast( int *__restrict__ dOutput, const int *__restrict__ dInput, const int srcLane){
    int val = dInput[threadIdx.x];
    // test1
    val = __shfl_sync(0xfff0fAfA, val, srcLane);
    // 特别地，各种函数返回的结果对被掩码排除的线程是没有定义的，不要尝试在这些被排除的线程中使用函数的返回值。
    unsigned int laneID = threadIdx.x & 0x1f;
    if(((1<<laneID) & 0xfff0fAfA)!=0)   // 取出该mask上 laneID处掩码值，1表示该线程参与运算; 1<<laneID 将1左移laneID位 置laneID位为1
        {dOutput[threadIdx.x] = val;
        printf("laneID:%d", laneID);
        }
    // mask:11111111,11110000,11111010,11111010
    //  如果不作判断，2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2
    // 0 2 0 2 2 2 2 2 ,0 2 0 2 2 2 2 2 ,0 0 0 0 2 2 2 2 ,2 2 2 2 2 2 2 2
    // test2
    //val = __shfl_sync(0xffffffff, val, srcLane, 16);
    // dOutput[threadIdx.x] = val;
    // mask:11111111,11111111,11111111,111111111
    // 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 18 18 18 18 18 18 18 18 18 18 18 18 18 18 18 18
    // test3
    //val = __shfl_sync(0xffffffff, val, srcLane, 32);
    // dOutput[threadIdx.x] = val;
    // mask:11111111,11111111,11111111,111111111
    // 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2 2
}


int main(){

    
    const int n = 32;
    int in[n];
    int out[n];
    int size = n * sizeof(int);

    std::cout <<"---------------input-------------" << std::endl;
    for (int i = 0; i < n; i++)
   {   
      in[i] = i;
      std::cout << in[i]<<" ";
   }
    std::cout <<"----------------------------" << std::endl;
   
    // GPU端分配内存
    int *in_d, *out_d;
    hipMalloc((void**)&in_d, size);
    hipMalloc((void**)&out_d, size);
    hipMemcpy(in_d, in, size, hipMemcpyHostToDevice); 
	// 核函数启动
	test_shuf_broadcast<<< 1, 32>>>(out_d, in_d, 2);
    
    hipMemcpy(out, out_d, size, hipMemcpyDeviceToHost);
    std::cout <<"---------------output-------------" << std::endl;
    for (int i = 0; i <     32; i++)
        std::cout << out[i]<<" ";
    std::cout <<"----------------------------" << std::endl;

        // 释放CPU端、GPU端的内存
    hipFree(in_d);
    hipFree(out_d);

}