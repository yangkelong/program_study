#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "config.h"

#include <optix.h>

#include "system_data.h"
#include "per_ray_data.h"
#include "vertex_attributes.h"
#include "function_indices.h"
#include "material_definition.h"
#include "light_definition.h"
#include "shader_common.h"
#include "transform.h"
#include "random_number_generators.h"


extern "C" __constant__ SystemData sysData;


extern "C" __global__ void __closesthit__radiance()
{
  GeometryInstanceData* theData = reinterpret_cast<GeometryInstanceData*>(optixGetSbtDataPointer());

  // Cast the hipDeviceptr_t to the actual format for Triangles geometry.
  const unsigned int thePrimitiveIndex = optixGetPrimitiveIndex();

  const uint3* indices = reinterpret_cast<uint3*>(theData->indices);
  const uint3  tri     = indices[thePrimitiveIndex];

  const TriangleAttributes* attributes = reinterpret_cast<TriangleAttributes*>(theData->attributes);

  const TriangleAttributes& attr0 = attributes[tri.x];
  const TriangleAttributes& attr1 = attributes[tri.y];
  const TriangleAttributes& attr2 = attributes[tri.z];

  const float2 theBarycentrics = optixGetTriangleBarycentrics(); // beta and gamma
  const float  alpha = 1.0f - theBarycentrics.x - theBarycentrics.y;

  const float3 ng = cross(attr1.vertex - attr0.vertex, attr2.vertex - attr0.vertex);
  const float3 tg = attr0.tangent * alpha + attr1.tangent * theBarycentrics.x + attr2.tangent * theBarycentrics.y;
  const float3 ns = attr0.normal  * alpha + attr1.normal  * theBarycentrics.x + attr2.normal  * theBarycentrics.y;
 
  // PERF This State lies in memory. It's more efficient to hold the data in registers.
  //      Problem is that more advanced material systems need the State all the time.
  State state; // All in world space coordinates!

  state.texcoord = attr0.texcoord * alpha + attr1.texcoord * theBarycentrics.x + attr2.texcoord * theBarycentrics.y;

  float4 objectToWorld[3];
  float4 worldToObject[3];

  getTransforms(optixGetTransformListHandle(0), objectToWorld, worldToObject); // Single instance level transformation list only.
  
  state.normalGeo = normalize(transformNormal(worldToObject, ng));
  state.tangent   = normalize(transformVector(objectToWorld, tg));
  state.normal    = normalize(transformNormal(worldToObject, ns));

  // Get the current rtPayload pointer from the unsigned int payload registers p0 and p1.
  PerRayData* thePrd = mergePointer(optixGetPayload_0(), optixGetPayload_1());

  thePrd->distance = optixGetRayTmax(); // Return the current path segment distance, needed for absorption calculations in the integrator.
  
  //thePrd->pos = optixGetWorldRayOrigin() + optixGetWorldRayDirection() * optixGetRayTmax();
  thePrd->pos += thePrd->wi * thePrd->distance; // DEBUG Check which version is more efficient.

  // Explicitly include edge-on cases as frontface condition!
  // Keeps the material stack from overflowing at silhouettes.
  // Prevents that silhouettes of thin-walled materials use the backface material.
  // Using the true geometry normal attribute as originally defined on the frontface!
  thePrd->flags |= (0.0f <= dot(thePrd->wo, state.normalGeo)) ? FLAG_FRONTFACE : 0;

  if ((thePrd->flags & FLAG_FRONTFACE) == 0) // Looking at the backface?
  {
    // Means geometric normal and shading normal are always defined on the side currently looked at.
    // This gives the backfaces of opaque BSDFs a defined result.
    state.normalGeo = -state.normalGeo;
    state.tangent   = -state.tangent;
    state.normal    = -state.normal;
    // Explicitly DO NOT recalculate the frontface condition!
  }
  
  thePrd->radiance = make_float3(0.0f);

  // When hitting a geometric light, evaluate the emission first, because this needs the previous diffuse hit's pdf.
  const int idLight = theData->idLight;

  if (0 <= idLight && (thePrd->flags & FLAG_FRONTFACE)) // This material is emissive and we're looking at the front face.
  {
    const float cosTheta = dot(thePrd->wo, state.normalGeo);
    if (DENOMINATOR_EPSILON < cosTheta)
    {
      const LightDefinition& light = sysData.lightDefinitions[idLight];

      float3 emission = make_float3(1.0f); // Neutral factor.
     
      if (light.textureEmission)
      {
        emission = make_float3(tex2D<float4>(light.textureEmission, state.texcoord.x, state.texcoord.y));
      }

      if (sysData.directLighting && (thePrd->flags & FLAG_DIFFUSE))
      {
        float pdfLight = (thePrd->distance * thePrd->distance) / (light.area * cosTheta);

        if (light.typeLight == TYPE_LIGHT_RECT && light.textureEmission)
        {
          pdfLight *= intensity(emission) / light.integral; // This must be the emission from the texture only!
        }
      
        // If it's an implicit light hit from a diffuse scattering event and the light emission (to be done with EDF evaluation) was not returning a zero pdf (e.g. backface or edge on).
        if (DENOMINATOR_EPSILON < pdfLight)
        {
          // Scale the emission with the heuristic between the previous diffuse BSDF sample pdf and this implicit light sample pdf.
          emission *= balanceHeuristic(thePrd->pdf, pdfLight);
        }
      }

      thePrd->radiance = emission * light.emission;
    }
  }

  // Start fresh with the next BSDF sample. (Either of these values remaining zero is an end-of-path condition.)
  // The pdf of the previous event was needed for the emission calculation above.
  thePrd->f_over_pdf = make_float3(0.0f);
  thePrd->pdf        = 0.0f;

  const MaterialDefinition& material = sysData.materialDefinitions[theData->idMaterial];

  // Only the last diffuse hit is tracked for multiple importance sampling of implicit light hits.
  thePrd->flags = (thePrd->flags & ~FLAG_DIFFUSE) | /* FLAG_HIT  | */ material.flags; // FLAG_THINWALLED can be set directly from the material.
 
  // PERF If the assigned material is the default black material (usually used on light geometry),
  // then end the path. No more radiance will be contributed by the following code.
  if (material.typeBXDF == TYPE_BXDF)
  {
    return;
  }

  state.albedo = material.albedo;

  if (material.textureAlbedo != 0)
  {
    const float3 texColor = make_float3(tex2D<float4>(material.textureAlbedo, state.texcoord.x, state.texcoord.y));

    // Modulate the incoming color with the texture.
    state.albedo *= texColor;               // linear color, resp. if the texture has been uint8 and readmode set to use sRGB, then sRGB.
    //state.albedo *= powf(texColor, 2.2f); // sRGB gamma correction done manually.
  }

  // Sample a new path direction. 
  const int callBXDF = NUM_LENS_TYPES + NUM_LIGHT_TYPES + material.typeBXDF * 2;

  optixDirectCall<void, const MaterialDefinition&, const State&, PerRayData*>(callBXDF, material, state, thePrd);

  if (sysData.directLighting == 0)
  {
    return;
  }

  // Direct lighting if the sampled BSDF was diffuse and any light is in the scene.
  const int numLights = sysData.numLights;
  if ((thePrd->flags & FLAG_DIFFUSE) && 0 < numLights)
  {
    // Sample one of many lights. 
    // The caller picks the light to sample. Make sure the index stays in the bounds of the sysData.lightDefinitions array.
    const int indexLight = (1 < numLights) ? clamp(static_cast<int>(floorf(rng(thePrd->seed) * numLights)), 0, numLights - 1) : 0;
    
    const LightDefinition& light = sysData.lightDefinitions[indexLight];
    
    const int callLight = NUM_LENS_TYPES + light.typeLight;

    LightSample lightSample = optixDirectCall<LightSample, const LightDefinition&, PerRayData*>(callLight, light, thePrd);

    if (0.0f < lightSample.pdf) // Useful light sample?
    {
      // Evaluate the BXDF in the light sample direction. Normally cheaper than shooting rays.
      // Returns BXDF f in .xyz and the BSDF pdf in .w
      // BXDF eval function is one index after the sample function.
      const float4 bxdf_pdf = optixDirectCall<float4, const MaterialDefinition&, const State&, const PerRayData* const, const float3>(callBXDF + 1, material, state, thePrd, lightSample.direction);

      if (0.0f < bxdf_pdf.w && isNotNull(make_float3(bxdf_pdf)))
      {
        // Pass the current payload registers through to the shadow ray.
        unsigned int p0 = optixGetPayload_0();
        unsigned int p1 = optixGetPayload_1();

        // Note that the sysData.sceneEpsilon is applied on both sides of the shadow ray [t_min, t_max] interval 
        // to prevent self-intersections with the actual light geometry in the scene.
        optixTrace(sysData.topObject,
                   thePrd->pos, lightSample.direction, // origin, direction
                   sysData.sceneEpsilon, lightSample.distance - sysData.sceneEpsilon, 0.0f, // tmin, tmax, time
                   OptixVisibilityMask(0xFF), OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT, // The shadow ray type only uses anyhit programs.
                   TYPE_RAY_SHADOW, NUM_RAY_TYPES, TYPE_RAY_SHADOW,
                   p0, p1); // Pass through thePrd to the shadow ray. It needs the seed and sets flags.

        if ((thePrd->flags & FLAG_SHADOW) == 0) // Shadow flag not set?
        {
          if (thePrd->flags & FLAG_VOLUME) // Supporting nested materials includes having lights inside a volume.
          {
            // Calculate the transmittance along the light sample's distance in case it's inside a volume.
            // The light must be in the same volume or it would have been shadowed.
            lightSample.emission *= expf(-lightSample.distance * thePrd->sigma_t);
          }

          if (TYPE_LIGHT_POINT <= light.typeLight)
          {
            // Singular light, cannot be hit implicitly, no light PDF and no MIS here.
            thePrd->radiance += make_float3(bxdf_pdf) * lightSample.emission * dot(lightSample.direction, state.normal);
          }
          else 
          {
            const float weightMis = balanceHeuristic(lightSample.pdf, bxdf_pdf.w);
            
            thePrd->radiance += make_float3(bxdf_pdf) * lightSample.emission * (weightMis * dot(lightSample.direction, state.normal) / lightSample.pdf);
          }
        }
      }
    }
  }
}
