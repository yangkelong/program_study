#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "config.h"

#include <optix.h>

#include "system_data.h"
#include "per_ray_data.h"
#include "vertex_attributes.h"
#include "function_indices.h"
#include "material_definition.h"
#include "light_definition.h"
#include "shader_common.h"
#include "transform.h"
#include "random_number_generators.h"


extern "C" __constant__ SystemData sysData;


extern "C" __global__ void __closesthit__edf_diffuse()
{
  GeometryInstanceData theData = sysData.geometryInstanceData[optixGetInstanceId()];

  // Cast the hipDeviceptr_t to the actual format for Triangles geometry.
  const unsigned int thePrimitiveIndex = optixGetPrimitiveIndex();

  const uint3* indices = reinterpret_cast<uint3*>(theData.indices);
  const uint3  tri     = indices[thePrimitiveIndex];

  const TriangleAttributes* attributes = reinterpret_cast<TriangleAttributes*>(theData.attributes);

  const TriangleAttributes& attr0 = attributes[tri.x];
  const TriangleAttributes& attr1 = attributes[tri.y];
  const TriangleAttributes& attr2 = attributes[tri.z];

  const float2 theBarycentrics = optixGetTriangleBarycentrics(); // beta and gamma
  const float  alpha = 1.0f - theBarycentrics.x - theBarycentrics.y;
  
  // PERF This State lies in memory. It's more efficient to hold the data in registers.
  //      Problem is that more advanced material systems need the State all the time.
  State state; // All in world space coordinates!

  state.normalGeo = cross(attr1.vertex - attr0.vertex, attr2.vertex - attr0.vertex);
  //state.tangent   = attr0.tangent * alpha + attr1.tangent * theBarycentrics.x + attr2.tangent * theBarycentrics.y; // PERF tangent is not used in this shader.
  state.normal    = attr0.normal  * alpha + attr1.normal  * theBarycentrics.x + attr2.normal  * theBarycentrics.y;
  state.texcoord  = attr0.texcoord * alpha + attr1.texcoord * theBarycentrics.x + attr2.texcoord * theBarycentrics.y;
  
  float4 objectToWorld[3];
  float4 worldToObject[3];

  getTransforms(optixGetTransformListHandle(0), objectToWorld, worldToObject); // Single instance level transformation list only.
  
  state.normalGeo = normalize(transformNormal(worldToObject, state.normalGeo));
  //state.tangent   = normalize(transformVector(objectToWorld, state.tangent));
  state.normal    = normalize(transformNormal(worldToObject, state.normal));

  // Get the current rtPayload pointer from the unsigned int payload registers p0 and p1.
  PerRayData* thePrd = mergePointer(optixGetPayload_0(), optixGetPayload_1());

  thePrd->distance = optixGetRayTmax(); // Return the current path segment distance, needed for absorption calculations in the integrator.
  
  //thePrd->pos = optixGetWorldRayOrigin() + optixGetWorldRayDirection() * optixGetRayTmax();
  thePrd->pos += thePrd->wi * thePrd->distance; // DEBUG Check which version is more efficient.

  // Explicitly include edge-on cases as frontface condition!
  // Keeps the material stack from overflowing at silhouettes.
  // Prevents that silhouettes of thin-walled materials use the backface material.
  // Using the true geometry normal attribute as originally defined on the frontface!
  thePrd->flags |= (0.0f <= dot(thePrd->wo, state.normalGeo)) ? FLAG_FRONTFACE : 0;

  if ((thePrd->flags & FLAG_FRONTFACE) == 0) // Looking at the backface?
  {
    // Means geometric normal and shading normal are always defined on the side currently looked at.
    // This gives the backfaces of opaque BSDFs a defined result.
    state.normalGeo = -state.normalGeo;
    //state.tangent   = -state.tangent;
    state.normal    = -state.normal;
    // Explicitly DO NOT recalculate the frontface condition!
  }
  
  thePrd->radiance = make_float3(0.0f);

  // When hitting a geometric light, evaluate the emission first, because this needs the previous diffuse hit's pdf.
  const int idLight = theData.idLight;

  if (0 <= idLight && (thePrd->flags & FLAG_FRONTFACE)) // This material is emissive and we're looking at the front face.
  {
    const float cosTheta = dot(thePrd->wo, state.normalGeo);
    if (DENOMINATOR_EPSILON < cosTheta)
    {
      const LightDefinition& light = sysData.lightDefinitions[idLight];

      float3 emission = make_float3(1.0f); // Neutral factor.
     
      if (light.textureEmission)
      {
        emission = make_float3(tex2D<float4>(light.textureEmission, state.texcoord.x, state.texcoord.y));
      }

      if (sysData.directLighting && (thePrd->flags & FLAG_DIFFUSE))
      {
        float pdfLight = (thePrd->distance * thePrd->distance) / (light.area * cosTheta);

        if (light.typeLight == TYPE_LIGHT_RECT && light.textureEmission)
        {
          pdfLight *= intensity(emission) / light.integral; // This must be the emission from the texture only!
        }
      
        // If it's an implicit light hit from a diffuse scattering event and 
        // the light emission was not returning a zero pdf (e.g. backface or edge on).
        // FIXME PERF The light emission pdf cannot be zero here because we're hitting the front face.
        // (Wouldn't matter for the balance heuristic anway, if b == 0.0 the result is always 1.0.)
        if (DENOMINATOR_EPSILON < pdfLight)
        {
          // Scale the emission with the heuristic between the previous diffuse BSDF sample pdf and this implicit light sample pdf.
          emission *= balanceHeuristic(thePrd->pdf, pdfLight);
        }
      }

      thePrd->radiance = emission * light.emission;
    }
  }

  // Start fresh with the next BSDF sample. (Either of these values remaining zero is an end-of-path condition.)
  // The pdf of the previous event was needed for the emission calculation above.
  thePrd->f_over_pdf = make_float3(0.0f);
  thePrd->pdf        = 0.0f;
  thePrd->flags     |= FLAG_TERMINATE;
}
