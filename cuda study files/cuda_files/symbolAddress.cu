
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
using namespace std;


__device__ int device_int;


__global__ void kernel()
{
    device_int = 1000;
}

int main()
{
    kernel<<<1, 1>>>();
    assert(hipGetLastError() == hipSuccess);

    int *host_pointer_to_device_int;
    /* Get a pointer to device_int. After this, I won't be able to access it,
     * but I'm going to copy its value with cudaMemcpy */
    hipGetSymbolAddress((void **)&host_pointer_to_device_int, HIP_SYMBOL(device_int));
    assert(hipGetLastError() == hipSuccess); // The above operation executed successfully

    int host_int;
    // Copy the device_int's value
    hipMemcpy(&host_int, host_pointer_to_device_int,
            sizeof(int), hipMemcpyDeviceToHost);

    // style2
    int value;
    hipMemcpyFromSymbol(&value, HIP_SYMBOL(device_int), hipMemcpyDeviceToHost);
    assert(hipGetLastError() == hipSuccess); // The above operation executed successfully
    cout << value << endl; // Everything's fine!
    cout << host_int << endl; // Everything's fine!
}

