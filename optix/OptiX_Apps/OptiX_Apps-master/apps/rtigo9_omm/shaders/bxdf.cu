#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "config.h"

#include <optix.h>

#include "per_ray_data.h"
#include "material_definition.h"
#include "vector_math.h"

// BXDF (default shader, black hole, end of path)

extern "C" __device__ void __direct_callable__sample_bxdf(const MaterialDefinition& material, const State& state, PerRayData* prd)
{
  // PERF Unused in RGP when FLAG_TERMINATE is set.
  prd->f_over_pdf = make_float3(0.0f); 
  prd->pdf        = 0.0f;

  prd->flags |= FLAG_TERMINATE;
}

// This function will be used for all specular materials.
// This is actually never reached in this simple material system, because the FLAG_DIFFUSE flag is not set when a specular BXDF is has been sampled.
extern "C" __device__ float4 __direct_callable__eval_bxdf(const MaterialDefinition& material, const State& state, const PerRayData* const prd, const float3 wiL)
{
  return make_float4(0.0f); // Specular.
}

